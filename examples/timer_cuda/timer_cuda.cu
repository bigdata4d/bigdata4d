#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <cstdio>

__global__ void compadd_kernel(float * __restrict__ a, float * __restrict__ b)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z;

  int ijk = i + j*384 + k*384*384;
  a[ijk] += b[ijk];
}

void prepareCUDA(float **a_gpu, float **b_gpu, float *a, float *b)
{
  int size = 384*384*384*sizeof(float);

  hipMalloc((void **)a_gpu, size);
  hipMalloc((void **)b_gpu, size);

  hipMemcpy(*a_gpu, a, size, hipMemcpyHostToDevice);
  hipMemcpy(*b_gpu, b, size, hipMemcpyHostToDevice);
}

void testCUDA(float *a_gpu, float *b_gpu)
{
  const int blocki = 32;
  const int blockj = 16;

  dim3 grid(384/blocki,384/blockj,384);
  dim3 block(blocki,blockj,1);

  compadd_kernel<<<grid, block>>>(a_gpu, b_gpu);
}

void waitCUDA()
{
  hipDeviceSynchronize();
}

void finishCUDA(float *a_gpu, float *b_gpu, float *a)
{
  int size = 384*384*384*sizeof(float);

  hipMemcpy(a, a_gpu, size, hipMemcpyDeviceToHost);

  hipFree(a_gpu);
  hipFree(b_gpu);
}

void testCUDA_thrust(thrust::device_vector<float> &a, thrust::device_vector<float> &b)
{
  thrust::transform(b.begin(), b.end(), a.begin(), a.begin(), thrust::plus<float>());
}
