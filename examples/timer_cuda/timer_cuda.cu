
#include <hip/hip_runtime.h>
__global__ void compadd_kernel(float *a, float *b)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z;
  int ijk = i + j*100 + k * 100*100;

  a[ijk] += b[ijk];
}

void prepareCUDA(float **a_gpu, float **b_gpu, float *a, float *b)
{
  int size = 100*100*100*sizeof(float);

  hipMalloc((void **)a_gpu, size);
  hipMalloc((void **)a_gpu, size);

  hipMemcpy(*a_gpu, a, size, hipMemcpyHostToDevice);
  hipMemcpy(*b_gpu, b, size, hipMemcpyHostToDevice);
}

void testCUDA(float *a_gpu, float *b_gpu)
{
  dim3 grid(100,100,100);
  dim3 block(1,1,1);

  compadd_kernel<<<grid, block>>>(a_gpu, b_gpu);
}

void waitCUDA()
{
  hipDeviceSynchronize();
}

void finishCUDA(float *a_gpu, float *b_gpu, float *a)
{
  int size = 100*100*100*sizeof(float);

  hipMemcpy(a, a_gpu, size, hipMemcpyDeviceToHost);

  hipFree(a_gpu);
  hipFree(b_gpu);
}
