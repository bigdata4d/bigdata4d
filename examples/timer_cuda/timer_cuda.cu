#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipblas.h>

__global__ void compadd_kernel(float * __restrict__ a, float * __restrict__ b, int n)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z;

  if(i < 384 && j < 384 && k < 384)
  {
    int ijk = i + j*384 + k*384*384;
    a[ijk] += b[ijk];
  }
}

void waitCUDA()
{
  hipDeviceSynchronize();
}

void prepareCUDA(float **a_gpu, float **b_gpu, float *a, float *b)
{
  int size = 384*384*384*sizeof(float);

  hipMalloc((void **)a_gpu, size);
  hipMalloc((void **)b_gpu, size);

  hipMemcpy(*a_gpu, a, size, hipMemcpyHostToDevice);
  hipMemcpy(*b_gpu, b, size, hipMemcpyHostToDevice);
}

void testCUDA(float *a_gpu, float *b_gpu)
{
  const int blocki = 256;
  const int blockj = 1;

  dim3 grid(384/blocki,384/blockj,384);
  dim3 block(blocki,blockj,1);

  compadd_kernel<<<grid, block>>>(a_gpu, b_gpu, 384);
}

void finishCUDA(float *a_gpu, float *b_gpu, float *a)
{
  int size = 384*384*384*sizeof(float);

  hipMemcpy(a, a_gpu, size, hipMemcpyDeviceToHost);

  hipFree(a_gpu);
  hipFree(b_gpu);
}

void testCUDA_thrust(thrust::device_vector<float> &a, const thrust::device_vector<float> &b)
{
  thrust::transform(b.begin(), b.end(), a.begin(), a.begin(), thrust::plus<float>());
}

void prepareCUDA_cublas(float **a_gpu, float **b_gpu, float *a, float *b)
{
  cublasInit();

  int size = 384*384*384*sizeof(float);

  hipMalloc((void **)a_gpu, size);
  hipMalloc((void **)b_gpu, size);

  hipblasSetVector(384*384*384, sizeof(float), a, 1, *a_gpu, 1);
  hipblasSetVector(384*384*384, sizeof(float), b, 1, *b_gpu, 1);
}

void testCUDA_cublas(float *a_gpu, float *b_gpu)
{
  hipblasSaxpy(384*384*384, 1., b_gpu, 1, a_gpu, 1);
}

void finishCUDA_cublas(float *a_gpu, float *b_gpu, float *a)
{
  hipblasGetVector(384*384*384, sizeof(float), a_gpu, 1, a, 1);
  cublasShutdown();

  hipFree(a_gpu);
  hipFree(b_gpu);
}
