#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Master.h"
#include "Grid.h"
#include "Field.h"
#include "Diffusion.h"
#include "DiffusionGPU.h"
#include "Timer.h"

int main(int argc, char *argv[])
{
  try
  {
    Master master;
    try
    {
      Grid<double> grid = createGrid<double>(master, 384, 256, 512, 3);

      Field<double,double> a  = createField<double>(master, grid, "a" );
      Field<double,double> at = createField<double>(master, grid, "at");

      a.randomize(10);

      Diffusion<double,double> diff(master, grid);

      Timer timer1(master, "Diffusion (CPU)");
      timer1.start();
      for(int n=0; n<10; ++n)
        diff.exec(at, a);
      timer1.end();

      DiffusionGPU<double,double> diffGPU(master, grid);

      thrust::device_vector<double> a_gpu (a.data.begin(), a.data.end());
      thrust::device_vector<double> at_gpu(a.data.size());

      Timer timer2(master, "Diffusion (GPU)");
      timer2.start();
      for(int n=0; n<10; ++n)
        diffGPU.exec(at_gpu, a_gpu);
      // wait until all calculations are done
      hipDeviceSynchronize();
      timer2.end();

      Field<double, double> at_cuda = createField<double>(master, grid, "at_cuda");
      thrust::copy(at_gpu.begin(), at_gpu.end(), at_cuda.data.begin());

      std::ostringstream message;
      message << "Elapsed time (s): "
              << std::setprecision(5) << timer1.getTotal() << ", "
              << std::setprecision(5) << timer2.getTotal() << ", "
              << "Speedup CUDA: " << timer1.getTotal() / timer2.getTotal() << "\n";
      master.printMessage(message.str());

      for(int n=3; n<at.data.size(); n+=384*384*20)
      {
        std::ostringstream message;
        message << std::setw(8);
        message << n << " = {" 
          << std::setw(6) << at.data[n] << ", "
          << std::setw(6) << at_cuda.data[n] << " }\n";
        master.printMessage(message.str());
      }
    }

    catch (std::exception &e)
    {
      std::ostringstream message;
      message << "Exception: " << e.what() << "\n";
      master.printMessage(message.str());
      throw 1;
    }
  }

  catch (...)
  {
    return 1;
  }

  return 0;
}
