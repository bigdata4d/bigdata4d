#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Master.h"
#include "Grid.h"
#include "Field.h"
#include "Diffusion.h"
#include "DiffusionGPU.h"
#include "Timer.h"

int main(int argc, char *argv[])
{
  try
  {
    Master master;
    try
    {
      Grid<double> grid = createGrid<double>(master, 2, 2, 2, 3);

      Field<double,double> a  = createField<double>(master, grid, "a" );
      Field<double,double> at = createField<double>(master, grid, "at");

      a.randomize(10);

      Diffusion<double,double> diff(master, grid);

      Timer timer1(master, "Diffusion (CPU)");
      timer1.start();
      for(int n=0; n<10; ++n)
        diff.exec(at, a);
      timer1.end();

      DiffusionGPU<double,double> diffGPU(master, grid);

      thrust::device_vector<double> a_gpu (a.data.begin(), a.data.end());
      thrust::device_vector<double> at_gpu(a.data.size());

      Timer timer2(master, "Diffusion (GPU)");
      timer2.start();
      for(int n=0; n<10; ++n)
        diffGPU.exec(at_gpu, a_gpu);
      // wait until all calculations are done
      hipDeviceSynchronize();
      timer2.end();

      Field<double, double> at_cuda = createField<double>(master, grid, "at_cuda");
      thrust::copy(at_gpu.begin(), at_gpu.end(), at_cuda.data.begin());

      std::ostringstream message;
      message << "Elapsed time (s): "
              << std::setprecision(5) << timer1.getTotal() << ", "
              << std::setprecision(5) << timer2.getTotal() << ", "
              << "Speedup CUDA: " << timer1.getTotal() / timer2.getTotal() << "\n";
      master.printMessage(message.str());

      // print some output
      const GridDims dims = grid.getDims();
      //for(long k=dims.kstart; k<dims.kend; ++k)
      //  for(long j=dims.jstart; j<dims.jend; ++j)
      //    for(long i=dims.istart; i<dims.iend; ++i)
      for(long k=0; k<dims.kcells; ++k)
        for(long j=0; j<dims.jcells; ++j)
          for(long i=0; i<dims.icells; ++i)
          {
            long ijk = i + j*dims.icells + k*dims.ijcells;
            std::ostringstream message;
            message << "(" << i <<  "," << j <<  "," << k << ") = " 
                    << at.data[ijk] << ", " << at_cuda.data[ijk] << "\n";
            master.printMessage(message.str());
          }
        }

    catch (std::exception &e)
    {
      std::ostringstream message;
      message << "Exception: " << e.what() << "\n";
      master.printMessage(message.str());
      throw 1;
    }
  }

  catch (...)
  {
    return 1;
  }

  return 0;
}
