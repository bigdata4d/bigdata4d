/*
 * BigDataGrid
 * Copyright (c) 2014-2015 Chiel van Heerwaarden
 *
 * Many of the classes and functions in BigDataGrid are derived from
 * MicroHH (https://github.com/MicroHH)
 *
 * This file is part of BigDataGrid
 *
 * BigDataGrid is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * BigDataGrid is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with BigDataGrid.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Master.h"
#include "Grid.h"
#include "Field.h"
#include "Diffusion.h"
#include "DiffusionGPU.h"
#include "Timer.h"

int main(int argc, char *argv[])
{
  try
  {
    try
    {
      Grid<double> grid = createGrid<double>(128, 128, 512, 3);

      Field<double,double> a  = createField<double>(grid, "a" );
      Field<double,double> at = createField<double>(grid, "at");

      a.randomize(10);

      Diffusion<double,double> diff(grid);

      Timer timer1("Diffusion (CPU)");
      timer1.start();
      for(int n=0; n<100; ++n)
        diff.exec(at, a);
      timer1.end();

      DiffusionGPU<double,double> diffGPU(grid);

      thrust::device_vector<double> a_gpu (a.data.begin(), a.data.end());
      thrust::device_vector<double> at_gpu(a.data.size());

      Timer timer2("Diffusion (GPU)");
      timer2.start();
      for(int n=0; n<100; ++n)
        diffGPU.exec(at_gpu, a_gpu);
      // wait until all calculations are done
      hipDeviceSynchronize();
      timer2.end();

      Field<double, double> at_cuda = createField<double>(grid, "at_cuda");
      thrust::copy(at_gpu.begin(), at_gpu.end(), at_cuda.data.begin());

      Master &master = Master::getInstance();
      std::ostringstream message;
      message << "Elapsed time (s): "
              << std::fixed << std::setprecision(5) << timer1.getTotal() << ", "
              << std::fixed << std::setprecision(5) << timer2.getTotal() << ", "
              << "Speedup CUDA: " << timer1.getTotal() / timer2.getTotal() << "\n";
      master.printMessage(message.str());

      // print some output
      const GridDims dims = grid.getDims();
      for(long k=dims.kstart+3; k<dims.kend; k+=dims.ktot/2)
        for(long j=dims.jstart+3; j<dims.jend; j+=dims.jtot/2)
          for(long i=dims.istart+3; i<dims.iend; i+=dims.itot/2)
          {
            long ijk = i + j*dims.icells + k*dims.ijcells;
            std::ostringstream message;
            message << "(" 
                    << std::setw(5) << i-dims.istart <<  "," 
                    << std::setw(5) << j-dims.jstart <<  "," 
                    << std::setw(5) << k-dims.kstart << ") = " 
                    << std::setw(10) << std::fixed << std::setprecision(3) << at.data[ijk] << ", "
                    << std::setw(10) << std::fixed << std::setprecision(3) << at_cuda.data[ijk] << "\n";
            master.printMessage(message.str());
          }
    }

    catch (std::exception &e)
    {
      std::ostringstream message;
      message << "Exception: " << e.what() << "\n";
      Master &master = Master::getInstance();
      master.printMessage(message.str());
      throw 1;
    }
  }

  catch (...)
  {
    return 1;
  }

  return 0;
}
