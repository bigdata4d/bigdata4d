#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "Master.h"
#include "Grid.h"
#include "Field.h"
#include "Diffusion.h"
#include "DiffusionGPU.h"
#include "Timer.h"

int main(int argc, char *argv[])
{
  try
  {
    Master master;
    try
    {
      Grid<double> grid = createGrid<double>(master, 128, 128, 512, 3);

      Field<double,double> a  = createField<double>(master, grid, "a" );
      Field<double,double> at = createField<double>(master, grid, "at");

      a.randomize(10);

      Diffusion<double,double> diff(master, grid);

      Timer timer1(master, "Diffusion (CPU)");
      timer1.start();
      for(int n=0; n<100; ++n)
        diff.exec(at, a);
      timer1.end();

      DiffusionGPU<double,double> diffGPU(master, grid);

      thrust::device_vector<double> a_gpu (a.data.begin(), a.data.end());
      thrust::device_vector<double> at_gpu(a.data.size());

      Timer timer2(master, "Diffusion (GPU)");
      timer2.start();
      for(int n=0; n<100; ++n)
        diffGPU.exec(at_gpu, a_gpu);
      // wait until all calculations are done
      hipDeviceSynchronize();
      timer2.end();

      Field<double, double> at_cuda = createField<double>(master, grid, "at_cuda");
      thrust::copy(at_gpu.begin(), at_gpu.end(), at_cuda.data.begin());

      std::ostringstream message;
      message << "Elapsed time (s): "
              << std::fixed << std::setprecision(5) << timer1.getTotal() << ", "
              << std::fixed << std::setprecision(5) << timer2.getTotal() << ", "
              << "Speedup CUDA: " << timer1.getTotal() / timer2.getTotal() << "\n";
      master.printMessage(message.str());

      // print some output
      const GridDims dims = grid.getDims();
      for(long k=dims.kstart+3; k<dims.kend; k+=dims.ktot/2)
        for(long j=dims.jstart+3; j<dims.jend; j+=dims.jtot/2)
          for(long i=dims.istart+3; i<dims.iend; i+=dims.itot/2)
          {
            long ijk = i + j*dims.icells + k*dims.ijcells;
            std::ostringstream message;
            message << "(" 
                    << std::setw(5) << i-dims.istart <<  "," 
                    << std::setw(5) << j-dims.jstart <<  "," 
                    << std::setw(5) << k-dims.kstart << ") = " 
                    << std::setw(10) << std::fixed << std::setprecision(3) << at.data[ijk] << ", "
                    << std::setw(10) << std::fixed << std::setprecision(3) << at_cuda.data[ijk] << "\n";
            master.printMessage(message.str());
          }
    }

    catch (std::exception &e)
    {
      std::ostringstream message;
      message << "Exception: " << e.what() << "\n";
      master.printMessage(message.str());
      throw 1;
    }
  }

  catch (...)
  {
    return 1;
  }

  return 0;
}
